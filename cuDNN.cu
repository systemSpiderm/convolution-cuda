#include <stdio.h>
#include <stdlib.h>
#include <hipDNN.h>

//从lower到upper的随机浮点数生成器
double get_random_double(double lower, double upper);

// 随机生成矩阵，100到100000
void initialize_matrix(double* mat, int size);


int main(int argc, char* argv[]) {

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int input_size, kernel_size = 3, stride, padding = 1;
    if (argc != 3) {
        fprintf(stderr, "The program %s did not get enough parameters, please enter input_size and stride\n", argv[0]);
        exit(1);
    }

    input_size = atoi(argv[1]);
    stride = atoi(argv[2]);

    // 初始化 cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // 输入和输出的维度
    int input_n = 1; 
    int input_c = 3; 
    int input_h = input_size;
    int input_w = input_size; 

    int kernel_n = 1;
    int kernel_c = 3;
    int kernel_h = kernel_size;
    int kernel_w = kernel_size; 

    int pad_h = padding;
    int pad_w = padding;
    int stride_h = stride;
    int stride_w = stride; 

    int output_n, output_c, output_h, output_w;

    // 创建输入和输出的张量描述符
    hipdnnTensorDescriptor_t input_descriptor;
    hipdnnTensorDescriptor_t output_descriptor;
    hipdnnFilterDescriptor_t kernel_descriptor;
    hipdnnConvolutionDescriptor_t convolution_descriptor;

    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnCreateFilterDescriptor(&kernel_descriptor);
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor);

    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE,
                                input_n, input_c, input_h, input_w);

    hipdnnSetFilter4dDescriptor(kernel_descriptor, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, 
                                kernel_n, kernel_c, kernel_h, kernel_w);

    hipdnnSetConvolution2dDescriptor(convolution_descriptor, pad_h, pad_w, stride_h, stride_w, 1, 1,
                                    HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_DOUBLE);

    // 获取输出张量的维度
    hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor, input_descriptor, kernel_descriptor,
                                            &output_n, &output_c, &output_h, &output_w);

    hipdnnSetTensor4dDescriptor(output_descriptor,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_DOUBLE,
                                output_n, output_c, output_h, output_w);

    // 分配内存
    size_t input_size_bytes = input_n * input_c * input_h * input_w * sizeof(double);
    size_t output_size_bytes = output_n * output_c * output_h * output_w * sizeof(double);
    size_t kernel_size_bytes = kernel_n * kernel_c * kernel_h * kernel_w * sizeof(double);

    double *d_input, *d_output, *d_kernel;
    hipMalloc(&d_input, input_size_bytes);
    hipMalloc(&d_output, output_size_bytes);
    hipMalloc(&d_kernel, kernel_size_bytes);

    // 初始化输入和卷积核
    double *h_input = (double*)malloc(input_size_bytes);
    double *h_kernel = (double*)malloc(kernel_size_bytes);

    initialize_matrix(h_input, input_n * input_c * input_h * input_w);
    initialize_matrix(h_kernel, kernel_n * kernel_c * kernel_h * kernel_w);

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);


    hipMemcpy(d_input, h_input, input_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, kernel_size_bytes, hipMemcpyHostToDevice);

    // 创建卷积前向算法描述符
    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    hipdnnGetConvolutionForwardAlgorithm(cudnn, input_descriptor, kernel_descriptor, 
                                        convolution_descriptor, output_descriptor, 
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convolution_algorithm);

    // 获取卷积前向操作的工作空间大小
    size_t workspace_bytes = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn,input_descriptor, kernel_descriptor,
                                            convolution_descriptor, output_descriptor,
                                            convolution_algorithm, &workspace_bytes);

    void* d_workspace = nullptr;
    hipMalloc(&d_workspace, workspace_bytes);

    // 执行卷积前向操作
    const double alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(cudnn,&alpha,input_descriptor,d_input,kernel_descriptor,
                            d_kernel,convolution_descriptor,convolution_algorithm,
                            d_workspace,workspace_bytes,&beta,output_descriptor,d_output);

    double *h_output = (double*)malloc(output_size_bytes);
    hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);

    // 释放资源
    free(h_input);
    free(h_kernel);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
    hipFree(d_workspace);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroy(cudnn);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("Elapsed time: %f ms\n", elapsedTime);


    return 0;
}


double get_random_double(double lower, double upper) {
    int random_int = rand();
    return lower + (double)(random_int / (RAND_MAX + 1.0)) * (upper - lower);
}

void initialize_matrix(double *matrix, int size) {
    for (int i = 0; i < size; i++) {
        matrix[i] = get_random_double(100.0, 100000.0);
    }
}